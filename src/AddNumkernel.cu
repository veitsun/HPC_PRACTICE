#include "hip/hip_runtime.h"
#include "AddNumkernel.cuh"

// 内核函数
__global__ void Add(int *a, int *b, int *c, int DX) {
  int f = blockIdx.x * blockDim.x + threadIdx.x;

  if (f >= DX)
    return;

  c[f] = a[f] + b[f];
}

// 调用内核函数
void AddKernel(int *a, int *b, int *c, int DX) {
  dim3 dimBlock = (128);
  dim3 dimGrid = ((DX + 128 - 1) / 128);
  Add<<<dimGrid, dimBlock>>>(a, b, c, DX);
  hipDeviceSynchronize();
}