#include "hip/hip_runtime.h"
// cuda stream
// cuda lib, cudnn cublas tensort
// stream-- 一系列的指令执行队列
// mul-stream -- asyn -- order-- asyn

#include <math.h>
#include <stdio.h>

// a[] + b[] = c[]

#define N (1024 * 1024)
#define FULL_SIZE (N * 30)
// stream[2]

__global__ void kernel(int *a, int *b, int *c) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < N) {
    int idx1 = (idx + 1) % 256;
    int idx2 = (idx + 2) % 256;
    float as = (a[idx] + a[idx1] + a[idx2]) / 3.0;
    float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0;

    c[idx] = (as + bs) / 2;
  }
}

int main() {
  hipDeviceProp_t prop;
  int whichDevice;
  hipGetDevice(&whichDevice);
  hipGetDeviceProperties(&prop, whichDevice);
  if (!prop.deviceOverlap) {
    printf("Your device will not support speed up from multi-streams\n");
    return 0;
  }

  hipEvent_t start, stop;
  float elapsedTime;

  hipStream_t my_stream[3];

  int *h_a, *h_b, *h_c;
  int *d_a0, *d_b0, *d_c0;
  int *d_a1, *d_b1, *d_c1;
  int *d_a2, *d_b2, *d_c2;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipStreamCreate(&my_stream[0]);
  hipStreamCreate(&my_stream[1]);
  hipStreamCreate(&my_stream[2]);

  hipMalloc((void **)&d_a0, N * sizeof(int));
  hipMalloc((void **)&d_b0, N * sizeof(int));
  hipMalloc((void **)&d_c0, N * sizeof(int));
  hipMalloc((void **)&d_a1, N * sizeof(int));
  hipMalloc((void **)&d_b1, N * sizeof(int));
  hipMalloc((void **)&d_c1, N * sizeof(int));
  hipMalloc((void **)&d_a2, N * sizeof(int));
  hipMalloc((void **)&d_b2, N * sizeof(int));
  hipMalloc((void **)&d_c2, N * sizeof(int));

  hipHostAlloc((void **)&h_a, FULL_SIZE * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&h_b, FULL_SIZE * sizeof(int), hipHostMallocDefault);
  hipHostAlloc((void **)&h_c, FULL_SIZE * sizeof(int), hipHostMallocDefault);

  for (int i = 0; i < FULL_SIZE; i++) {
    h_a[i] = rand() % 1024;
    h_b[i] = rand() % 1024;
  }

  hipEventRecord(start);
  for (int i = 0; i < FULL_SIZE; i += N * 1) {
    hipMemcpyAsync(d_a0, h_a + i, N * sizeof(int), hipMemcpyHostToDevice,
                    my_stream[0]);
    // hipMemcpyAsync(d_a1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice,
    // my_stream[1]); hipMemcpyAsync(d_a2, h_a+i+N+N, N*sizeof(int),
    // hipMemcpyHostToDevice, my_stream[2]);
    hipMemcpyAsync(d_b0, h_a + i, N * sizeof(int), hipMemcpyHostToDevice,
                    my_stream[0]);
    // hipMemcpyAsync(d_b1, h_a+i+N, N*sizeof(int), hipMemcpyHostToDevice,
    // my_stream[1]); hipMemcpyAsync(d_b2, h_a+i+N+N, N*sizeof(int),
    // hipMemcpyHostToDevice, my_stream[2]);

    kernel<<<N / 256, 256, 0, my_stream[0]>>>(d_a0, d_b0, d_c0);
    // kernel<<<N/256, 256, 0, my_stream[1]>>>(d_a1, d_b1, d_c1);
    // kernel<<<N/256, 256, 0, my_stream[2]>>>(d_a2, d_b2, d_c2);

    hipMemcpyAsync(h_c + i, d_c0, N * sizeof(int), hipMemcpyDeviceToHost,
                    my_stream[0]);
    // hipMemcpyAsync(h_c+i+N, d_c0, N*sizeof(int), hipMemcpyDeviceToHost,
    // my_stream[0]); hipMemcpyAsync(h_c+i+N+N, d_c0, N*sizeof(int),
    // hipMemcpyDeviceToHost, my_stream[0]);
  }

  hipStreamSynchronize(my_stream[0]);
  hipStreamSynchronize(my_stream[1]);
  hipStreamSynchronize(my_stream[2]);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&elapsedTime, start, stop);

  printf("Time: %3.2f ms\n", elapsedTime);

  // hipFree

  return 0;
}
