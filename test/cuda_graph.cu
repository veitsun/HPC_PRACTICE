#include "hip/hip_runtime.h"
#include "Timer.cuh"
#include "utils.cuh"
// #include <__clang_cuda_builtin_vars.h>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#define N 500000

__global__ void shortKernel(float *out_d, float *in_d) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N) {
    out_d[idx] = in_d[idx] * 2.0f;
  }
}

#define NSTEP 1000
#define NKERNEL 20

const int thread_per_block = 256;
const int block_per_grid = (N + thread_per_block - 1) / thread_per_block;

int main() {
  float *out_d = nullptr;
  float *in_d = nullptr;
  hipMalloc(&out_d, N * sizeof(float));
  hipMalloc(&in_d, N * sizeof(float));
  const float valueA = 0.1;
  const float valueB = 0.5;
  // // 两种初始化形式
  // init_two_array_mem_by_cpu(out_d, in_d, N, valueA,
  //                            valueB); // 方式一，用 CPU 进行初始化

  hipDeviceSynchronize();

  init_two_array_mem_by_gpu<<<block_per_grid, thread_per_block>>>(
      out_d, in_d, N, valueA, valueB); // 方式二，用 GPU 进行初始化

  hipDeviceSynchronize();
  // hipStream_t stream;
  // hipError_t err = hipStreamCreate(&stream);
  // if (err != hipSuccess) {
  //   fprintf(stderr, "create stream failed : %s\n", hipGetErrorString(err));
  // }
  startTimer();
  for (int istep = 0; istep < NSTEP; ++istep) {
    for (int ikernel = 0; ikernel < NKERNEL; ++ikernel) {
      shortKernel<<<block_per_grid, thread_per_block, 0>>>(out_d, in_d);
      // hipStreamSynchronize(stream); // 等待这个流完成
    }
  }
  // hipStreamDestroy(stream); // 用完 stream 后销毁

  hipDeviceSynchronize();

  float times = stopTimer();
  printf("程序内核所执行的时间 %.2f ms\n", times);
  printf("每个内核所执行的时间 %.2f ms\n", (times / NSTEP * NKERNEL));

  hipFree(out_d);
  hipFree(in_d);
  return 0;
}