#include <hip/hip_runtime.h>
#include <iostream>

// CUDA 内核函数，进行简单的向量加法
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

int main() {
  // 定义向量大小
  int N = 1 << 20; // 1M 个元素
  size_t size = N * sizeof(float);

  // 在主机 (Host) 上分配内存
  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);
  float *h_C = (float *)malloc(size);

  // 初始化向量 A 和 B
  for (int i = 0; i < N; ++i) {
    h_A[i] = static_cast<float>(i);
    h_B[i] = static_cast<float>(i * 2);
  }

  // 在设备 (Device) 上分配内存
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  // 创建两个 CUDA Stream
  hipStream_t stream1, stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  // 设置线程和块的数量
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  // 使用 Stream1：异步将 A 数组从主机传输到设备
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream1);

  // 使用 Stream2：异步将 B 数组从主机传输到设备
  hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream2);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time;
  hipEventRecord(start, 0);
  // 使用 Stream1：在设备上启动向量加法内核
  vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A, d_B, d_C, N);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("Vector_add_stream Time elapsed %f ms\n", time);

  // 使用 Stream1：异步将结果 C 从设备传输回主机
  hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream1);

  // 等待所有 Stream 执行完成
  hipStreamSynchronize(stream1);
  hipStreamSynchronize(stream2);

  // 检查结果
  bool success = true;
  for (int i = 0; i < N; ++i) {
    if (fabs(h_C[i] - (h_A[i] + h_B[i])) > 1e-5) {
      success = false;
      break;
    }
  }
  if (success) {
    std::cout << "Vector addition completed successfully!" << std::endl;
  } else {
    std::cout << "Vector addition failed!" << std::endl;
  }

  // 释放资源
  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(h_A);
  free(h_B);
  free(h_C);

  return 0;
}
