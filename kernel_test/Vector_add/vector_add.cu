// #include <algorithm>
// #include <__clang_cuda_builtin_vars.h>

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>

__global__ void add(float *a, float *b, float *c, int N) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  const int N = 1 << 20;
  float *hostA;
  float *hostB;
  float *hostC;
  hostA = (float *)malloc(sizeof(float) * N);
  hostB = (float *)malloc(sizeof(float) * N);
  hostC = (float *)malloc(sizeof(float) * N);
  for (int i = 0; i < N; ++i) {
    hostA[i] = static_cast<float>(i);
    hostB[i] = static_cast<float>(i * 2);
  }
  float *deviceA, *deviceB, *deviceC;
  hipMalloc((void **)&deviceA, N * sizeof(int));
  hipMalloc((void **)&deviceB, sizeof(int) * N);
  hipMalloc((void **)&deviceC, sizeof(int) * N);
  hipMemcpy(deviceA, hostA, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(deviceC, hostC, N * sizeof(int), hipMemcpyHostToDevice);
  // 执行内核函数
  // 设置线程和块的数量
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time;
  hipEventRecord(start, 0);

  add<<<blocksPerGrid, threadsPerBlock>>>(deviceA, deviceB, deviceC, N);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("Vector_add Time elapsed %f ms\n", time);

  // cudaMemcpy(hostC, deviceC, N * sizeof(int), cudaMemcpyDeviceToHost);
  // for (int i = 0; i < N; i++) {
  //   std::cout << hostC[i] << " ";
  // }
  std::cout << std::endl;
  free(hostA);
  free(hostB);
  free(hostC);
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);
  return 0;
}