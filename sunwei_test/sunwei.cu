
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void add(int *a, int *b, int *c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}
