#include "hip/hip_runtime.h"
#include "include/CInitialData.h"
// #include "include/CPrintMatrix.h"
#include "include/Num.h"
#include "include/common.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hipblas.h>
// #include <iostream>
using namespace std;

__global__ void MulMatrixOnDevice(int M, int N, int K, float alpha, float *A,
                                  float *B, float beta, float *C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < M && col < N) {
    float temp = 0.0;
    for (int k = 0; k < K; k++) {
      temp += A[row * K + k] * B[k * N + col];
    }
    C[row * N + col] = alpha * temp + beta * C[row * N + col];
  }
}

int main(int argc, char **argv) {
  float *hostA;
  float *hostB;
  float *hostC;
  float *gpuRef;

  float alpha = 1.0;
  float beta = 1.0;

  // 给主机上的三个矩阵分配内存
  hostA = (float *)malloc(elemNum * sizeof(float));
  hostB = (float *)malloc(elemNum * sizeof(float));
  hostC = (float *)malloc(elemNum * sizeof(float));
  gpuRef = (float *)malloc(elemNum * sizeof(float));

  // 主机上的三个矩阵初始化数据
  CInitialData cinitialData;
  cinitialData.initialDataABC(hostA, hostB, hostC, nx, ny);
  memset(gpuRef, 0, elemNum * sizeof(float));

  // cout << "测试主机上的三个矩阵是否已经被初始化数据" << endl;
  // CPrintMatrix cprintmatrix;
  // cprintmatrix.printMatrixABC(hostA, hostB, hostC, nx, ny);

  // -------------------------------------------------------------------------------------GPU计时

  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // -----------------------------------------------------------------------------------------
  // 使用cuda kernel 来执行矩阵乘法
  dim3 blockDim(BLOCK_DIM_x, BLOCK_DIM_y);
  dim3 gridDim((ny + blockDim.x - 1) / blockDim.x,
               (nx + blockDim.y - 1) / blockDim.y);
  float *deviceA;
  float *deviceB;
  float *deviceC;
  CHECK(hipMalloc((float **)&deviceA, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceB, elemNum * sizeof(float)));
  CHECK(hipMalloc((float **)&deviceC, elemNum * sizeof(float)));
  CHECK(hipMemcpy(deviceA, hostA, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceB, hostB, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceC, hostC, elemNum * sizeof(float),
                   hipMemcpyHostToDevice));
  hipEventRecord(start, 0);
  MulMatrixOnDevice<<<gridDim, blockDim>>>(nx, nx, nx, alpha, deviceA, deviceB,
                                           beta, deviceC);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  hipEventElapsedTime(&time, start, stop);
  printf("MulMatrixOnDevice Time elapsed %f ms\n", time);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  CHECK(hipMemcpy(gpuRef, deviceC, elemNum * sizeof(float),
                   hipMemcpyDeviceToHost));
  CHECK(hipDeviceSynchronize());
  // -----------------------------------------------------------------------------------------
  CHECK(hipFree(deviceA));
  CHECK(hipFree(deviceB));
  CHECK(hipFree(deviceC));
  free(hostA);
  free(hostB);
  free(hostC);
  free(gpuRef);
  return 0;
}