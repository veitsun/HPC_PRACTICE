#include "include/CInitialData.h"
#include "include/CPrintMatrix.h"
#include "include/Num.h"
#include "include/common.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <hipblas.h>
#include <iostream>
using namespace std;
// ---------------------------------------------------------------------------cublas
void matMult_cublas(int M, int N, int K, float alpha, float *A, float *B,
                    float beta, float *C, hipblasHandle_t cuHandle,
                    float *cublasRef) {
  float *cublasdeviceA;
  float *cublasdeviceB;
  float *cublasdeviceC;
  hipEvent_t start, stop;
  float time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // 在显存中为计算矩阵开辟空间
  CHECK(hipMalloc((void **)&cublasdeviceA, elemNum * sizeof(float)));
  CHECK(hipMalloc((void **)&cublasdeviceB, elemNum * sizeof(float)));
  CHECK(hipMalloc((void **)&cublasdeviceC, elemNum * sizeof(float)));
  // 将主机上的数据拷贝到设备中
  hipblasSetVector(elemNum, sizeof(float), A, 1, cublasdeviceA, 1);
  hipblasSetVector(elemNum, sizeof(float), B, 1, cublasdeviceB, 1);
  hipblasSetVector(elemNum, sizeof(float), C, 1, cublasdeviceC, 1);
  // 传递矩阵相乘中的参数，并执行内核函数，矩阵相乘
  // hipblasSgemm(cuHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
  //             cublasdeviceA, N, cublasdeviceB, K, &beta, cublasdeviceC, M);
  hipEventRecord(start, 0);
  // ------------------------------------------------------------------------------------------------
  // int repeat = 20;
  // for (int i = 0; i < repeat; i++) {
  hipblasSgemm(cuHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
              cublasdeviceA, N, cublasdeviceB, K, &beta, cublasdeviceC, M);
  // }

  // ---------------------------------------------------------------------------------------------------
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);
  printf("matMult_cublas Time elapsed %f ms\n", time);
  hipblasGetVector(elemNum, sizeof(float), cublasdeviceC, 1, cublasRef, 1);
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(cublasdeviceA);
  hipFree(cublasdeviceB);
  hipFree(cublasdeviceC);
}

int main(int argc, char **argv) {
  float *hostA;
  float *hostB;
  float *hostC;
  float *cublasRef;
  float alpha = 1.0;
  float beta = 0.0;
  // 给主机上的三个矩阵分配内存
  hostA = (float *)malloc(elemNum * sizeof(float));
  hostB = (float *)malloc(elemNum * sizeof(float));
  hostC = (float *)malloc(elemNum * sizeof(float));
  cublasRef = (float *)malloc(elemNum * sizeof(float));
  // 主机上的三个矩阵初始化数据
  CInitialData cinitialData;
  cinitialData.initialDataABCByFile(hostA, hostB, hostC, n, n);
  memset(cublasRef, 0, elemNum * sizeof(float));

  // cout << "测试主机上的三个矩阵是否已经被初始化数据" << endl;
  CPrintMatrix cprintmatrix;
  // cprintmatrix.printMatrixABC(hostA, hostB, hostC, nx, ny);
  // -----------------------------------------------------------------------------------------
  cout << "使用cublas 执行矩阵乘法" << endl;
  // 使用cublas 执行矩阵乘法
  // 创建并初始化cublas对象
  // 若是cublas对象在主函数中初始化，cublas方法在其他函数中调用，需要将cuHandle传入该函数，并在函数内创建status对象
  hipblasHandle_t cuHandle;
  hipblasStatus_t status = hipblasCreate(&cuHandle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
      cout << "cublas 对象实例化出错" << endl;
    }
    getchar();
    return EXIT_FAILURE;
  }
  matMult_cublas(n, n, n, alpha, hostA, hostB, beta, hostC, cuHandle,
                 cublasRef);
  hipblasDestroy(cuHandle);
  // -----------------------------------------------------------------
  // cprintmatrix.printMatrixCinFile(cublasRef, nx, ny);
  cprintmatrix.printMatrixCinFile(cublasRef, n, n);
  // cprintmatrix.printMatrix(hostC, nx, ny);

  free(hostA);
  free(hostB);
  free(hostC);
  return 0;
}